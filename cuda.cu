#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 512;


int n_body;
int n_iteration;

double total_time;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] += vx[i]*dt;
        y[i] += vy[i]*dt;

        double r = sqrt(radius2);
        if (x[i] <= r){
            x[i] = r + err;
            vx[i] = -vx[i];
        }
        else if (x[i] >= bound_x-r){
            x[i] = bound_x - r - err;
            vx[i] = -vx[i];
        }
        if (y[i] <= r){
            y[i] = r + err;
            vy[i] = -vy[i];
        }
        else if (y[i] >= bound_y-r){
            y[i] = bound_y - r - err;
            vy[i] = -vy[i];
        } // handle wall collision
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double x0 = x[i];
        double y0 = y[i];
        double vx0 = vx[i];
        double vy0 = vy[i];
        double ax = 0.0;
        double ay = 0.0;

        for (int j=0; j<n; j++){
            if (i == j) continue;
            double delta_x = x[i] - x[j];
            double delta_y = y[i] - y[j];
            double dist_s = delta_x*delta_x + delta_y*delta_y;
            bool isCollision = false;
            if (dist_s <= radius2*4){
                dist_s = radius2*4;
                isCollision = true;
            }
            double dist = sqrt(dist_s);

            if (isCollision) {
                double dot_prod = delta_x * (vx[i] - vx[j]) + delta_y * (vy[i] - vy[j]);
                double value = 2 / (m[i] + m[j]) * dot_prod / dist_s;
                vx0 -= value * delta_x * m[j];
                vy0 -= value * delta_y * m[j];
                
                x0 += delta_x / dist * sqrt(radius2) / 2.0;
                y0 += delta_y / dist * sqrt(radius2) / 2.0;
            } else {
                double F = m[i]*m[j]*gravity_const / dist_s;
                ax -= F * delta_x / m[i];
                ay -= F * delta_y / m[i];
            }
        }
        vx[i] += ax * dt;
        vy[i] += ay * dt;

        double r = sqrt(radius2);
        if (x[i] <= r){
            x[i] = r + err;
            vx[i] = -vx[i];
        }
        else if (x[i] >= bound_x-r){
            x[i] = bound_x - r - err;
            vx[i] = -vx[i];
        }
        if (y[i] <= r){
            y[i] = r + err;
            vy[i] = -vy[i];
        }
        else if (y[i] >= bound_y-r){
            y[i] = bound_y - r - err;
            vy[i] = -vy[i];
        }
    }
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}


void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);
        total_time += time_span.count();

        l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    total_time = 0.0;
    master();

    printf("Student ID: 119010437\n"); // replace it with your student id
    printf("Name: ZHANG Shiyi\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");
    printf("Total running time: %.4f\n",total_time);

    return 0;

}


